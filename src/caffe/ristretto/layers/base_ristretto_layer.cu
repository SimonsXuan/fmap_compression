#include "hip/hip_runtime.h"
#include "ristretto/base_ristretto_layer.hpp"
#include "ristretto/base_ristretto_layer.cuh"

namespace caffe {

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeWeights_gpu(
      vector<shared_ptr<Blob<Dtype> > > weights_quantized, const int rounding,
      const bool bias_term) {
  Dtype* weight = weights_quantized[0]->mutable_gpu_data();
  const int cnt_weight = weights_quantized[0]->count();
  switch (precision_) {
  case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
    Trim2FixedPoint_gpu(weight, cnt_weight, bw_params_, rounding, fl_params_);
    if (bias_term) {
      Trim2FixedPoint_gpu(weights_quantized[1]->mutable_gpu_data(),
          weights_quantized[1]->count(), bw_params_ + bw_layer_out_, rounding, bw_params_ + fl_layer_out_);
    }
    break;
  default:
    LOG(FATAL) << "Unknown trimming mode: " << precision_;
    break;
  }
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeLayerInputs_gpu(
    Dtype* data, const int count) {
  switch (precision_) {
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, bw_layer_in_, rounding_, fl_layer_in_);
      break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << precision_;
      break;
  }
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeLayerOutputs_gpu(Dtype* data,
      const int count) {
  switch (precision_) {
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, bw_layer_out_, rounding_, fl_layer_out_);
      break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << precision_;
      break;
  }
}

template <typename Dtype>
__global__ void Trim2FixedPoint_kernel(Dtype* data, const int cnt,
      const int bit_width, const int rounding, const int fl) {
	CUDA_KERNEL_LOOP(index, cnt) {
    // round data
    data[index] *= powf(2, fl);
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      data[index] = rintf(data[index]);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      //data[index] += RandUniform_device(index);
      data[index] = __float2int_rd(data[index] + RandUniform_device(index)); // ??? somehow this is working ???
      break;
    default:
      break;
    }
    // saturate data
    Dtype max_data = (powf(2, bit_width - 1) - 1.0);
    Dtype min_data = -powf(2, bit_width - 1);
    data[index] = fmaxf(fminf(data[index], max_data), min_data);
    // back to float
    data[index] *= powf(2, -fl);
	}
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2FixedPoint_gpu(Dtype* data, const int cnt,
      const int bit_width, const int rounding, const int fl) {
  Trim2FixedPoint_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bit_width, rounding, fl);
}

// Explicit instantiations
template void BaseRistrettoLayer<double>::QuantizeWeights_gpu(
    vector<shared_ptr<Blob<double> > > weights_quantized, const int rounding,
    const bool bias_term);
template void BaseRistrettoLayer<float>::QuantizeWeights_gpu(
    vector<shared_ptr<Blob<float> > > weights_quantized, const int rounding,
    const bool bias_term);
template void BaseRistrettoLayer<double>::QuantizeLayerInputs_gpu(double* data,
    const int count);
template void BaseRistrettoLayer<float>::QuantizeLayerInputs_gpu(float* data,
    const int count);
template void BaseRistrettoLayer<double>::QuantizeLayerOutputs_gpu(
    double* top_data, const int top_count);
template void BaseRistrettoLayer<float>::QuantizeLayerOutputs_gpu(
    float* top_data, const int top_count);
template void BaseRistrettoLayer<double>::Trim2FixedPoint_gpu(double* data,
    const int cnt, const int bit_width, const int rounding, const int fl);
template void BaseRistrettoLayer<float>::Trim2FixedPoint_gpu(float* data,
    const int cnt, const int bit_width, const int rounding, const int fl);

}  // namespace caffe
