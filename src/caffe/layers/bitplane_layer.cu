#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/bitplane_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void forward_i2b(const int n, const Dtype* in, Dtype* out, const int b, const int fl) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype otmp = in[index] * powf(2, fl);
    unsigned utmp = __float2uint_rz(otmp);
    unsigned btmp = (utmp >> b) & 0x00000001;
    out[index] = __uint2float_rz(btmp);
  }
}

/*template <typename Dtype>
__global__ void forward_b2i(const int n, const Dtype* in, Dtype* out, const Dtype scale) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] += in[index] * scale;
  }
}*/

template <typename Dtype>
void BitplaneLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  //
  const int num      = bottom[0]->num(); // batches
  const int channels = bottom[0]->channels();
  const int height   = bottom[0]->height();
  const int width    = bottom[0]->width();
  const int spatial  = height*width;
  const int fmap     = channels*spatial;
  const int count    = num*fmap;
  //
  const bool dir = this->layer_param_.bitplane_param().direction();
  const int bw   = this->layer_param_.bitplane_param().bw_layer();
  const int fl   = this->layer_param_.bitplane_param().fl_layer();
  //
  const int fmapI    = fmap/bw;
  const int countI   = count/bw;
  // set to zero
  if (dir != true) {
    caffe_gpu_set(countI, Dtype(0), top_data);
  }
  //
  for (int n = 0; n < num; ++n) {
    for (int b = 0; b < bw; ++b) {
      if (dir == true) { // int to bits
        forward_i2b<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
            <<<CAFFE_GET_BLOCKS(fmap), CAFFE_CUDA_NUM_THREADS>>>(
            fmap, bottom_data + n*fmap, top_data + fmap*(b + n*bw), b, fl);
      } else { // bits to int
        const Dtype scale = powf(2, b-fl); // forward scaler
        caffe_gpu_axpy(fmapI, scale, bottom_data + fmapI*(b + n*bw), top_data + n*fmapI);
        /*forward_b2i<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
            <<<CAFFE_GET_BLOCKS(fmapI), CAFFE_CUDA_NUM_THREADS>>>(
            fmapI, bottom_data + fmapI*(b + n*bw), top_data + n*fmapI, scale);*/
      }
    }
  }
  //
  CUDA_POST_KERNEL_CHECK;
  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

template <typename Dtype>
__global__ void backward_i2b(const int n, const Dtype* in, Dtype* out, const Dtype* sw) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (sw[index] > Dtype(0));
  }
}

/*template <typename Dtype>
__global__ void backward_b2i(const int n, const Dtype* in, Dtype* out, const Dtype scale) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] += in[index] * scale;
  }
}*/

template <typename Dtype>
void BitplaneLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    //
    const int num      = bottom[0]->num(); // batches
    const int channels = bottom[0]->channels();
    const int height   = bottom[0]->height();
    const int width    = bottom[0]->width();
    const int spatial  = height*width;
    const int fmap     = channels*spatial;
    const int count    = num*fmap;
    //
    const bool dir = this->layer_param_.bitplane_param().direction();
    const int bw   = this->layer_param_.bitplane_param().bw_layer();
    const int fl   = this->layer_param_.bitplane_param().fl_layer();
    //
    const int fmapI    = fmap/bw;
    //const int countI   = count/bw;
    const Dtype scale = 2.0 * powf(bw, -1); // simplified gradient scaler
    //
    if (dir == true) { // set to zero
      caffe_gpu_set(count, Dtype(0), bottom_diff);
    }
    //
    for (int n = 0; n < num; ++n) {
      for (int b = 0; b < bw; ++b) {
        if (dir != true) { // int to bits
          backward_i2b<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
              <<<CAFFE_GET_BLOCKS(fmapI), CAFFE_CUDA_NUM_THREADS>>>(
              fmapI, top_diff + n*fmapI, bottom_diff + fmapI*(b + n*bw), bottom_data + fmapI*(b + n*bw));
        } else { // bits to int
          caffe_gpu_axpy(fmap, scale, top_diff + fmap*(b + n*bw), bottom_diff + n*fmap);
          /*backward_b2i<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
              <<<CAFFE_GET_BLOCKS(fmap), CAFFE_CUDA_NUM_THREADS>>>(
              fmap, top_diff + fmap*(b + n*bw), bottom_diff + n*fmap, scale);*/
        }
      }
    }
    //
    CUDA_POST_KERNEL_CHECK;
    // << " count: " << count << " bottom_data: "
    //     << (unsigned long)bottom_data
    //     << " top_data: " << (unsigned long)top_data
    //     << " blocks: " << CAFFE_GET_BLOCKS(count)
    //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BitplaneLayer);

}  // namespace caffe
